#include "utils.cuh"

void _cudaCheck(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("[CUDA ERROR] at file %s(line %d):\n%s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    return;
}

void randomize_matrix(float *mat, int N) {
    std::random_device rd;  
    std::mt19937 gen(rd()); // 使用随机设备初始化生成器  

    // 创建一个在[0, 2000)之间均匀分布的分布对象  
    std::uniform_int_distribution<> dis(0, 2000); 
    for (int i = 0; i < N; i++) {
        // 生成随机数，限制范围在[-1.0,1.0]
        mat[i] = (dis(gen)-1000)/1000.0;  
    }
}

void print_matrix(float* a, int M, int N) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%7.3f", a[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}
