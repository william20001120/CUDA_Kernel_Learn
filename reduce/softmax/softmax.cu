#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <float.h>
#include "utils.cuh"

const int N = 2048;
constexpr size_t BLOCK_SIZE = 256;
const int repeat_times = 10;

__global__ void setToNegativeMax(float* d_value) {
    *d_value = -FLT_MAX;
}

__device__ static float atomicMax(float* address, float val) {
	int* address_as_i = (int*)address;  // address转为int指针
	int old = *address_as_i;  // address中的旧值，用int解码
	int assumed;
	do {
		assumed = old;  // assumed存储旧值
		old = atomicCAS(address_as_i, assumed, __float_as_int(fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__global__ void max_kernel(float* input, float* output, int N) {
    __shared__ float s_mem[32];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadIdx.x / warpSize;
    int laneId = threadIdx.x % warpSize;

    // 求M(max)
    float val = (idx < N) ? input[idx] : (-FLT_MAX);
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    if (laneId == 0) s_mem[warpId] = val;
    __syncthreads();

	if (warpId == 0) {
        int warpNum = blockDim.x / warpSize;
        val = (laneId < warpNum) ? s_mem[laneId] : (-FLT_MAX);
        for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
            val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
        }
        if (laneId == 0) atomicMax(output, val);
    }
}


__global__ void sum_kernel(float* input, float* sum, float* max_val, int N) {
    __shared__ float s_mem[32];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int warpId = threadIdx.x / warpSize;
    int laneId = threadIdx.x % warpSize;

    float val = (idx < N) ? expf(input[idx] - *max_val) : 0.0f;
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    if (laneId == 0) s_mem[warpId] = val;
    __syncthreads();

    if (warpId == 0) {
        int warpNum = blockDim.x / warpSize;
        val = (laneId < warpNum) ? s_mem[laneId] : 0.0f;
        for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        }
        if (laneId == 0) atomicAdd(sum, val);
    }
}


__global__ void softmax_kernel(float* input, float* output, float* sum, float* max_val, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) output[idx] = expf(input[idx] - *max_val) / (*sum);
}

void softmax(float* input, float* output, int N, float* M, float* sum) {
    *M = *(std::max_element(input, input + N));  // 计算输入数组的最大值
    *sum = 0;
    for (int i = 0; i < N; i++) {
        output[i] = std::exp(input[i] - *M);  // 每个数先减去最大值，再求exp，避免溢出
        *sum += output[i];
    }
    for (int i = 0; i < N; i++) {
        output[i] /= *sum;
    }
}


void call_softmax_v1(float* output, float* input_device, float* output_device, float* total_device, float* total_max_device, int N) {
    int block_size = BLOCK_SIZE;
    int grid_size  = CEIL(N, BLOCK_SIZE);

    // 1. 初始化
    cudaCheck(hipMemset(total_device, 0, sizeof(float)));      // total需要设置为0
    cudaCheck(hipMemset(total_max_device, 0, sizeof(float)));
    
    // 2. 计算和
    sum_kernel<<<grid_size, block_size>>>(input_device, total_device, total_max_device, N);

    // 3. 计算softmax (没有减去最大值)
    softmax_kernel<<<grid_size, block_size>>>(input_device, output_device, total_device, total_max_device, N);
}


void call_softmax_v2(float* output, float* input_device, float* output_device, float* total_device, float* total_max_device, int N) {
    int block_size = BLOCK_SIZE;
    int grid_size  = CEIL(N, BLOCK_SIZE);

    // 1. 初始化
    cudaCheck(hipMemset(total_device, 0, sizeof(float)));  // total需要设置为0
    setToNegativeMax<<<1,1>>>(total_max_device);            // total_max_device设置为最小FLOAT值

    // 2. 计算最大值
    max_kernel<<<grid_size, block_size>>>(input_device, total_max_device, N);

    // 3. 计算和
    sum_kernel<<<grid_size, block_size>>>(input_device, total_device, total_max_device, N);

    // 4. 计算softmax (减去最大值避免溢出)
    softmax_kernel<<<grid_size, block_size>>>(input_device, output_device, total_device, total_max_device, N);
}


int main() {
    float* input  = (float*)malloc(sizeof(float) * N);
    float* output_ref = (float*)malloc(sizeof(float) * N);
    float* M = (float*)malloc(sizeof(float));
    float* sum = (float*)malloc(sizeof(float));
    for (int i = 0; i < N; i++) {
        input[i] = i/(float)N;
    }
    float total_time_h = TIME_RECORD(repeat_times, ([&]{softmax(input, output_ref, N, M, sum);}));
    printf("[softmax_cpu]: total_time_h = %f ms\n", total_time_h / repeat_times);

    float* input_device  = nullptr;
    float* output_device = nullptr;
    float* total_device = nullptr;
    float* total_max_device = nullptr;
    cudaCheck(hipMalloc(&input_device, N * sizeof(float)));
    cudaCheck(hipMalloc(&output_device, N * sizeof(float)));
    cudaCheck(hipMalloc(&total_device, 1 * sizeof(float)));
    cudaCheck(hipMalloc(&total_max_device, 1 * sizeof(float)));

    cudaCheck(hipMemcpy(input_device, input, N * sizeof(float), hipMemcpyHostToDevice));
    float* output = (float*)malloc(sizeof(float) * N);

    // softmax_v1
    float total_time_1 = TIME_RECORD(repeat_times, ([&]{call_softmax_v1(output, input_device, output_device, total_device, total_max_device, N);}));
    printf("[softmax_kernel1]: total_time_1 = %f ms\n", total_time_1 / repeat_times);
    cudaCheck(hipMemcpy(output, output_device, N * sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize(); 
    verify_matrix(output, output_ref, N);

    // softmax_v2
    float total_time_2 = TIME_RECORD(repeat_times, ([&]{call_softmax_v2(output, input_device, output_device, total_device, total_max_device, N);}));
    printf("[softmax_kernel2]: total_time_2 = %f ms\n", total_time_2 / repeat_times);
    cudaCheck(hipMemcpy(output, output_device, N * sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    verify_matrix(output, output_ref, N);

    float* total_host = (float*)malloc(sizeof(float));
    float* total_max_host = (float*)malloc(sizeof(float));
    cudaCheck(hipMemcpy(total_host, total_device, sizeof(float), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(total_max_host, total_max_device, sizeof(float), hipMemcpyDeviceToHost));

    free(input);
    free(output);
    free(M);
    free(sum);
    free(output_ref);
    cudaCheck(hipFree(input_device));
    cudaCheck(hipFree(output_device));
    cudaCheck(hipFree(total_device));
    cudaCheck(hipFree(total_max_device));
    return 0;
}