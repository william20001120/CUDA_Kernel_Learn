#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <float.h>
#include "utils.cuh"

// cpu: 计算每行的softmax
void softmax_row(float* input, float* output, int M, int N) {
    for (int row = 0; row < M; row++) {
        // 第row行
        float* input_tmp  = input + row * N;
        float* output_tmp = output + row * N;
        float max_val = *(std::max_element(input_tmp, input_tmp + N));  // 计算输入数组的最大值
        float sum = 0;
        for (int i = 0; i < N; i++) {
            output_tmp[i] = std::exp(input_tmp[i] - max_val);  // 每个数先减去最大值，再求exp，避免溢出
            sum += output_tmp[i];
        }
        for (int i = 0; i < N; i++) {
            output_tmp[i] /= sum;
        }
    }
}

// cpu: 计算每列的softmax
void softmax_col(float* x, float* y, int M, int N) {
    for (int col = 0; col < N; col++) {
        // 偏移到当前列
        float* x_col = x + col;
        float* y_col = y + col;

        // 计算当前列的最大值、和
        float max_val = -FLT_MAX;
        for (int i = 0; i < M; i++) {
            max_val = max(x_col[i*N], max_val);
        }
        float sum = 0;
        for (int i = 0; i < M; i++) {
            sum += exp(x_col[i*N] - max_val);
        }
        for (int i = 0; i < M; i++) {
            y_col[i*N] = exp(x_col[i*N] - max_val) / sum;
        }
    }
}

// gpu: 计算每行的softmax
__global__ void softmax_row_kernel(float* input, float* output, int M, int N) {
    __shared__ float s_max_val;
    __shared__ float s_sum;
    int laneId = threadIdx.x % warpSize;
    // 当前行
    int row = blockIdx.x;
    if (row >= M) return;

    int iteration = CEIL(N, warpSize);  // 每个线程负责计算的数据个数

    // 求每一行最大值
    float max_val = -FLT_MAX;
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int col = i * warpSize + laneId;
        max_val = (col < N) ? fmaxf(max_val, input[row * N + col]) : max_val;
    }
    #pragma unroll
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        max_val = fmaxf(max_val, __shfl_down_sync(0xFFFFFFFF, max_val, offset));
    }
    if (laneId == 0) s_max_val = max_val;  // 最大值汇总到第一个线程，第一个线程将它搬运到s_mem

    // 求每一行的和，且要减去最大值
    float sum = 0.0f;
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int col = i * warpSize + laneId;
        sum += (col < N) ? expf(input[row * N + col] - s_max_val) : 0.0f;
    }
    #pragma unroll
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }
    if (laneId == 0) s_sum = sum;  // sum值汇总到第一个线程，第一个线程将它搬运到s_mem

    // 计算每一行的softmax
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int col = i * warpSize + laneId;
        if (col < N) output[row * N + col] = expf(input[row * N + col] - s_max_val) / s_sum;
    }
}

// gpu: 计算每行的softmax, 改用 __shfl_xor_sync 后, 每个线程的
// 寄存器的 max_val 和 sum 都是最终的结果，就不用写到共享内存再读取了
__global__ void softmax_row_kernel2(float* input, float* output, int M, int N) {
    int laneId = threadIdx.x % warpSize;
    // 当前行
    int row = blockIdx.x;
    if (row >= M) return;

    int iteration = CEIL(N, warpSize);  // 每个线程负责计算的数据个数

    // 求每一行最大值
    float max_val = -FLT_MAX;
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int col = i * warpSize + laneId;
        max_val = (col < N) ? fmaxf(max_val, input[row * N + col]) : max_val;
    }
    #pragma unroll
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        max_val = fmaxf(max_val, __shfl_xor_sync(0xFFFFFFFF, max_val, offset));
    }

    // 求每一行的和，且要减去最大值
    float sum = 0.0f;
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int col = i * warpSize + laneId;
        sum += (col < N) ? expf(input[row * N + col] - max_val) : 0.0f;
    }
    #pragma unroll
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        sum += __shfl_xor_sync(0xFFFFFFFF, sum, offset);
    }

    // 计算每一行的softmax
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int col = i * warpSize + laneId;
        if (col < N) output[row * N + col] = expf(input[row * N + col] - max_val) / sum;
    }
}

// gpu: 计算每列的softmax
__global__ void softmax_col_kernel(float* input, float* output, int M, int N) {
    __shared__ float s_max_val;
    __shared__ float s_sum;
    int laneId = threadIdx.x % warpSize;
    // 当前列
    int col = blockIdx.x;
    if (col >= N) return;

    int iteration = CEIL(M, warpSize);  // 每个线程负责计算的数据个数

    // 求每一列最大值
    float max_val = -FLT_MAX;
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int row = i * warpSize + laneId;
        max_val = (row < M) ? fmaxf(max_val, input[row * N + col]) : max_val;
    }
    #pragma unroll
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        max_val = fmaxf(max_val, __shfl_down_sync(0xFFFFFFFF, max_val, offset));
    }
    if (laneId == 0) s_max_val = max_val;  // 最大值汇总到第一个线程，第一个线程将它搬运到s_mem

    // 求每一列的和，且要减去最大值
    float sum = 0.0f;
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int row = i * warpSize + laneId;
        sum += (row < M) ? expf(input[row * N + col] - s_max_val) : 0.0f;
    }
    #pragma unroll
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }
    if (laneId == 0) s_sum = sum;  // sum值汇总到第一个线程，第一个线程将它搬运到s_mem

    // 计算每一列的softmax
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int row = i * warpSize + laneId;
        if (row < M) output[row * N + col] = expf(input[row * N + col] - s_max_val) / s_sum;
    }
}

// gpu: 计算每列的softmax, 改用 __shfl_xor_sync 后，每个线程的
// 寄存器的 max_val 和 sum 都是最终的结果，就不用写到共享内存再读取了
__global__ void softmax_col_kernel2(float* input, float* output, int M, int N) {
    int laneId = threadIdx.x % warpSize;
    // 当前列
    int col = blockIdx.x;
    if (col >= N) return;

    int iteration = CEIL(M, warpSize);  // 每个线程负责计算的数据个数

    // 求每一列最大值
    float max_val = -FLT_MAX;
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int row = i * warpSize + laneId;
        max_val = (row < M) ? fmaxf(max_val, input[row * N + col]) : max_val;
    }
    #pragma unroll
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        max_val = fmaxf(max_val, __shfl_xor_sync(0xFFFFFFFF, max_val, offset));
    }

    // 求每一列的和，且要减去最大值
    float sum = 0.0f;
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int row = i * warpSize + laneId;
        sum += (row < M) ? expf(input[row * N + col] - max_val) : 0.0f;
    }
    #pragma unroll
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        sum += __shfl_xor_sync(0xFFFFFFFF, sum, offset);
    }

    // 计算每一列的softmax
    #pragma unroll
    for (int i = 0; i < iteration; i++) {
        int row = i * warpSize + laneId;
        if (row < M) output[row * N + col] = expf(input[row * N + col] - max_val) / sum;
    }
}


int main() {
    const int M = 2048;
    const int N = 64;
    const int repeat_times = 10;

    float* input      = (float*)malloc(M * N * sizeof(float));  // 输入是M*N的矩阵
    float* output     = (float*)malloc(M * N * sizeof(float));  // 输出是M*N的矩阵
    float* output_ref = (float*)malloc(M * N * sizeof(float));  // 输出是M*N的矩阵(cpu)

    // 初始化输入
    randomize_matrix(input, M*N);

    // cpu, 计算一行的softmax
    float total_time_h = TIME_RECORD(repeat_times, ([&]{softmax_row(input, output_ref, M, N);}));
    printf("[softmax_row_cpu]: total_time_h = %f ms\n", total_time_h / repeat_times);

    float* input_device  = nullptr;
    float* output_device = nullptr;
    cudaCheck(hipMalloc(&input_device,  M * N * sizeof(float)));
    cudaCheck(hipMalloc(&output_device, M * N * sizeof(float)));
    cudaCheck(hipMemcpy(input_device, input, M * N * sizeof(float), hipMemcpyHostToDevice));

    // gpu, 计算一行的softmax
    float total_time_d = TIME_RECORD(repeat_times, ([&]{softmax_row_kernel2<<<M, 32>>>(input_device, output_device, M, N);}));
    printf("[softmax_row_gpu]: total_time_d = %f ms\n", total_time_d / repeat_times);
    cudaCheck(hipMemcpy(output, output_device, M * N * sizeof(float), hipMemcpyDeviceToHost));
    verify_matrix(output, output_ref, M*N);

    // cpu, 计算一列的softmax
    float total_time_h2 = TIME_RECORD(repeat_times, ([&]{softmax_col(input, output_ref, M, N);}));
    printf("[softmax_col_cpu]: total_time_h = %f ms\n", total_time_h2 / repeat_times);

    // gpu, 计算一列行的softmax
    float total_time_d2 = TIME_RECORD(repeat_times, ([&]{softmax_col_kernel2<<<N, 32>>>(input_device, output_device, M, N);}));
    printf("[softmax_col_gpu]: total_time_d = %f ms\n", total_time_d2 / repeat_times);
    cudaCheck(hipMemcpy(output, output_device, M * N * sizeof(float), hipMemcpyDeviceToHost));
    verify_matrix(output, output_ref, M*N);

    free(input);
    free(output);
    free(output_ref);
    cudaCheck(hipFree(input_device));
    cudaCheck(hipFree(output_device));
    return 0;
}