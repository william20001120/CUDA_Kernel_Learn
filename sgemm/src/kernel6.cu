#include "hip/hip_runtime.h"
#include "kernel6.cuh"

template<const int BM,
         const int BN,
         const int BK,
         const int TM,
         const int TN>
__global__ void sgemm_v6(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    const int block_row_thread = BN / TN;
    const int block_col_thread = BM / TM;
    const int thread_num = block_row_thread * block_col_thread; // 一个线程负责计算block中TM*TN个元素

    // 当前线程对应thread tile的左上角元素在block中的位置
    int tx = (threadIdx.x % block_row_thread) * TN;
    int ty = (threadIdx.x / block_row_thread) * TM;

    __shared__ float As[BK * BM];
    __shared__ float Bs[BK * BN];


    const int ldg_a_num = BK * BM / thread_num / 4; // 每个线程搬运4个浮点数，完成搬运至As需要每个线程搬运ldg_a_num轮
    const int ldg_b_num = BK * BN / thread_num / 4; // 每个线程搬运4个浮点数，完成搬运至Bs需要每个线程搬运ldg_b_num轮

    int a_tile_row = threadIdx.x / (BK / 4); // 每行4个浮点数作为一个内存块，当前线程负责第a_tile_row行的第a_tile_col个内存块的搬运
    int a_tile_col = threadIdx.x % (BK / 4) * 4;
    int a_tile_stride = BM / ldg_a_num; // 一共BM行，搬运ldg_a_num轮，每轮需要搬运a_tile_stride行

    int b_tile_row = threadIdx.x / (BN / 4); // 每行4个浮点数作为一个内存块，当前线程负责第b_tile_row行的第b_tile_col个内存块的搬运
    int b_tile_col = threadIdx.x % (BN / 4) * 4;
    int b_tile_stride = BK / ldg_b_num; // 一共BK行，搬运ldg_b_num轮，每轮需要搬运b_tile_stride行

    float accum[TM][TN] = {0.}; // 每个线程负责TM*TN个元素，则需要申请TM*TN个寄存器保存累加值

    // 计算ldg_a_num的所有参数必须全部是const，否则不能用来申明数组大小
    float ldg_a_reg[4 * ldg_a_num] = {0.}; // 每个线程搬运ldg_a_num轮，寄存器缓存ldg_a_num个float4元素，用于转置As矩阵

    float a_frag[TM];  // 缓存As共享内存
    float b_frag[TN];  // 缓存Bs共享内存

    // 移动到当前block
    A = &A[by * BM * K];
    B = &B[bx * BN];
    C = &C[by * BM * N + bx * BN];

#pragma unroll
    for (int k = 0; k < K; k += BK) {
#pragma unroll
        for (int i = 0; i < BM; i += a_tile_stride) {
            int ldg_index = i / a_tile_stride * 4;  // 第ldg_index轮
            FETCH_FLOAT4(ldg_a_reg[ldg_index]) =
                    FETCH_FLOAT4(A[OFFSET(a_tile_row + i, a_tile_col, K)]);
            // As转置存，其中ldg_a_reg做中间缓存，目的是读取时可以按FLOAT4读取
            As[OFFSET(a_tile_col, i + a_tile_row, BM)] = ldg_a_reg[ldg_index];
            As[OFFSET(a_tile_col + 1, i + a_tile_row, BM)] = ldg_a_reg[ldg_index + 1];
            As[OFFSET(a_tile_col + 2, i + a_tile_row, BM)] = ldg_a_reg[ldg_index + 2];
            As[OFFSET(a_tile_col + 3, i + a_tile_row, BM)] = ldg_a_reg[ldg_index + 3];
        }
#pragma unroll
        for (int i = 0; i < BK; i += b_tile_stride) {
            FETCH_FLOAT4(Bs[OFFSET(b_tile_row + i, b_tile_col, BN)]) =
                    FETCH_FLOAT4(B[OFFSET(b_tile_row + i, b_tile_col, N)]); // 不需要转置
        }
        __syncthreads();
        A += BK;
        B += BK * N;
#pragma unroll
        for (int i = 0; i < BK; i++) {
#pragma unroll
            for (int m = 0; m < TM; m += 4) {
                FETCH_FLOAT4(a_frag[m]) = FETCH_FLOAT4(As[OFFSET(i, ty + m, BM)]); // 偏移到当前thread tile
            }
#pragma unroll
            for (int n = 0; n < TN; n += 4) {
                FETCH_FLOAT4(b_frag[n]) = FETCH_FLOAT4(Bs[OFFSET(i, tx + n, BN)]); // 偏移到当前thread tile
            }
#pragma unroll
            for (int m = 0; m < TM; m++) {
#pragma unroll
                for (int n = 0; n < TN; n++) {
                    accum[m][n] += a_frag[m] * b_frag[n];
                }
            }
        }
        __syncthreads();
    }
#pragma unroll
    for (int m = 0; m < TM; m++) {
#pragma unroll
        for (int n = 0; n < TN; n += 4) {
            float4 ctmp = FETCH_FLOAT4(C[OFFSET(ty + m, tx + n, N)]);
            //float4 atmp = FETCH_FLOAT4(accum[m][n]);
            ctmp.x = alpha * accum[m][n] + beta * ctmp.x;
            ctmp.y = alpha * accum[m][n + 1] + beta * ctmp.y;
            ctmp.z = alpha * accum[m][n + 2] + beta * ctmp.z;
            ctmp.w = alpha * accum[m][n + 3] + beta * ctmp.w;
            FETCH_FLOAT4(C[OFFSET(ty + m, tx + n, N)]) = ctmp;
        }
    }
}

// template instantiation declaration
template __global__ void sgemm_v6<128, 128, 8, 8, 8>(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);
