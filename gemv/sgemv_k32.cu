#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h> 

#define CEIL(a,b) ((a)+((b)-1))/(b)
#define checkCudaErrors(func) {                                                   \
    hipError_t e = (func);                                                       \
    if(e != hipSuccess)                                                          \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));  \
}

// dim3 dimGrid(M);
// dim3 dimBlock(32);
// 适合K∈[32,128]使用，小于32或大于128有进一步的优化方法
__global__ void sgemv_k32(float* A, float* x, float* y, int M, int K) {
    int laneId = threadIdx.x % warpSize;
    int row = blockIdx.x;  // 0~M-1
    if (row >= M) return;

    float res = 0.0f;
    int kIteration = CEIL(K, warpSize);  // 每个线程需要负责计算的数据个数

    #pragma unroll
    for(int i = 0; i < kIteration; i++){
        int col = i * warpSize + laneId;
        res += (col < K) ? A[row * K + col] * x[col] : 0.0f;
    }

    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        res += __shfl_down_sync(0xFFFFFFFF, res, offset);
    }

    if(laneId == 0) y[row] = res;
}

int main() {
    size_t M = 1024;
    size_t K = 32;

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_x = sizeof(float) * K;
    size_t bytes_y = sizeof(float) * M;
    float* h_A  = (float*)malloc(bytes_A);
    float* h_x  = (float*)malloc(bytes_x);
    float* h_y  = (float*)malloc(bytes_y);
    float* h_y1 = (float*)malloc(bytes_y);

    float* d_A;
    float* d_x;
    float* d_y;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_x, bytes_x));
    checkCudaErrors(hipMalloc(&d_y, bytes_y));

    double duration[2] = {0, 0};
    double GFLOPS[2] = {0, 0};
    double GFLOPs = 2.0 * M * 1 * K;

    // 生成A的数据
    for( int i = 0; i < M * K; i++ ) {
        h_A[i] = (float)i/K;
    }

    // 生成x的数据
    for( int i = 0; i < K; i++ ) {
        h_x[i] = 1;
    }
    memset(h_y,  0, M * sizeof(float));
    memset(h_y1, 0, M * sizeof(float));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int iteration = 1000;

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_x, h_x, bytes_x, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_y, h_y, bytes_y, hipMemcpyHostToDevice));
    
    checkCudaErrors(hipEventRecord(start));

    for (int run = 0 ; run < iteration; run ++ ) {
        dim3 dimGrid(M);
        dim3 dimBlock(32);
        sgemv_k32<<<dimGrid, dimBlock>>>(d_A, d_x, d_y, M, K);
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    checkCudaErrors(hipMemcpy( h_y, d_y, bytes_y, hipMemcpyDeviceToHost));

    duration[0] = msecTotal / iteration;
    GFLOPS[0] = (GFLOPs * 1.0e-9f) / (duration[0] / 1000.0f);
    printf( "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        GFLOPS[0],
        duration[0],
        GFLOPs);

    // cublas
    hipblasHandle_t blas_handle;  
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy( d_y, h_y1, bytes_y, hipMemcpyHostToDevice));

    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < iteration; run ++ ) {
        hipblasSgemv (blas_handle, HIPBLAS_OP_T, 
            K, M, &alpha, 
            d_A, K, d_x, 1, &beta, d_y, 1
        );
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_y1, d_y, bytes_y, hipMemcpyDeviceToHost));

    duration[1] = msecTotal / iteration;
    GFLOPS[1] = (GFLOPs * 1.0e-9f) / (duration[1] / 1000.0f);
    printf( "CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        GFLOPS[1],
        duration[1],
        GFLOPs);

    hipblasDestroy(blas_handle);
    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M; i++) {
        double abs_err = fabs(h_y[i] - h_y1[i]);
        double dot_length = M;
        double abs_val = fabs(h_y[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_y[i], h_y1[i], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    
    free(h_A);
    free(h_x);
    free(h_y);
    free(h_y1);
}
