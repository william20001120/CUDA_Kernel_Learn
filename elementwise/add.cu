#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define FLOAT4(a) *(float4*)(&(a))
#define CEIL(a,b) ((a+b-1)/(b))
#define cudaCheck(err) _cudaCheck(err, __FILE__, __LINE__)
void _cudaCheck(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("[CUDA ERROR] at file %s(line %d):\n%s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    return;
};


__global__ void elementwise_add_float4(float* a, float* b, float* c, int N) {
    int idx = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
    if (idx >= N) return;
    
    float4 tmp_a = FLOAT4(a[idx]);
    float4 tmp_b = FLOAT4(b[idx]);
    float4 tmp_c;
    tmp_c.x = tmp_a.x + tmp_b.x;
    tmp_c.y = tmp_a.y + tmp_b.y;
    tmp_c.z = tmp_a.z + tmp_b.z;
    tmp_c.w = tmp_a.w + tmp_b.w;
    FLOAT4(c[idx]) = tmp_c;
}

int main() {
    constexpr int N = 7;
    float* a_h = (float*)malloc(N * sizeof(float));
    float* b_h = (float*)malloc(N * sizeof(float));
    float* c_h = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        a_h[i] = i;
        b_h[i] = N-1-i;
    }

    float* a_d = nullptr;
    float* b_d = nullptr;
    float* c_d = nullptr;
    cudaCheck(hipMalloc((void**)&a_d, N * sizeof(float)));
    cudaCheck(hipMalloc((void**)&b_d, N * sizeof(float)));
    cudaCheck(hipMalloc((void**)&c_d, N * sizeof(float)));
    cudaCheck(hipMemcpy(a_d, a_h, N * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(b_d, b_h, N * sizeof(float), hipMemcpyHostToDevice));

    int block_size = 1024;
    int grid_size = CEIL(CEIL(N,4), 1024);
    elementwise_add_float4<<<grid_size, block_size>>>(a_d, b_d, c_d, N);

    cudaCheck(hipMemcpy(c_h, c_d, N * sizeof(float), hipMemcpyDeviceToHost));
    printf("a_h:\n");
    for (int i = 0; i < N; i++ ) {
        if (i == N-1) printf("%f\n", a_h[i]);
        else printf("%f ", a_h[i]);
    }
    printf("b_h:\n");
    for (int i = 0; i < N; i++ ) {
        if (i == N-1) printf("%f\n", b_h[i]);
        else printf("%f ", b_h[i]);
    }
    printf("c_h:\n");
    for (int i = 0; i < N; i++ ) {
        if (i == N-1) printf("%f\n", c_h[i]);
        else printf("%f ", c_h[i]);
    }
    return 0;
}